#include "hip/hip_runtime.h"
// pipeline_gpu.cu
// Enhanced full GPU pipeline with detailed performance and classification export.
// Build: nvcc -std=c++14 -O2 -arch=sm_61 -o pipeline_gpu pipeline_gpu.cu

#include "cuda_utils.h"
#include <fstream>
#include <iostream>
#include <chrono>
#include <vector>
#include <algorithm>

__global__ void conv_naive(const float* input, const float* weight, const float* bias,
                           float* output, int Kin, int Kout, int H, int W, int K) {
    int out_h = H - K + 1;
    int out_w = W - K + 1;
    int ox = blockIdx.x * blockDim.x + threadIdx.x;
    int oy = blockIdx.y * blockDim.y + threadIdx.y;
    int oc = blockIdx.z;
    if (ox >= out_w || oy >= out_h) return;

    float val = bias[oc];
    for (int ic = 0; ic < Kin; ++ic)
        for (int ky = 0; ky < K; ++ky)
            for (int kx = 0; kx < K; ++kx) {
                int in_y = oy + ky;
                int in_x = ox + kx;
                int in_idx = (ic * H + in_y) * W + in_x;
                int w_idx = ((oc * Kin + ic) * K + ky) * K + kx;
                val += input[in_idx] * weight[w_idx];
            }
    output[(oc * out_h + oy) * out_w + ox] = fmaxf(val, 0.0f);
}

__global__ void maxpool_sample(const float* input, float* output, int C, int H, int W, int K) {
    int out_w = W / K, out_h = H / K;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int c = blockIdx.y;
    if (idx >= out_h * out_w) return;
    int oy = idx / out_w, ox = idx % out_w;
    float best = -1e9f;
    for (int ky = 0; ky < K; ++ky)
        for (int kx = 0; kx < K; ++kx) {
            int in_y = oy * K + ky, in_x = ox * K + kx;
            float v = input[(c * H + in_y) * W + in_x];
            if (v > best) best = v;
        }
    output[(c * out_h + oy) * out_w + ox] = best;
}

int main() {
    try {
        int n_images;
        auto images_u8 = load_mnist_images("../data/t10k-images-idx3-ubyte", n_images);
        auto labels = load_mnist_labels("../data/t10k-labels-idx1-ubyte", n_images);

        std::vector<int> conv_meta, fc_meta;
        auto conv_w = load_csv_weights("../exports/pipeline/conv_weight.csv", conv_meta);
        auto conv_b = load_csv_weights("../exports/pipeline/conv_bias.csv", conv_meta);
        auto fc_w = load_csv_weights("../exports/pipeline/fc_weight.csv", fc_meta);
        auto fc_b = load_csv_weights("../exports/pipeline/fc_bias.csv", fc_meta);

        int Kout = conv_meta[0], Kin = conv_meta[1], K = conv_meta[2];
        int H = 28, W = 28;
        int out_h = H - K + 1, out_w = W - K + 1;
        int pool_k = 2, pool_h = out_h / pool_k, pool_w = out_w / pool_k;
        int D = Kout * pool_h * pool_w, out_dim = fc_meta[0];

        float *d_input, *d_conv_out, *d_pool, *d_conv_w, *d_conv_b;
        hipMalloc(&d_input, Kin * H * W * sizeof(float));
        hipMalloc(&d_conv_out, Kout * out_h * out_w * sizeof(float));
        hipMalloc(&d_pool, Kout * pool_h * pool_w * sizeof(float));
        hipMalloc(&d_conv_w, conv_w.size() * sizeof(float));
        hipMalloc(&d_conv_b, conv_b.size() * sizeof(float));

        hipMemcpy(d_conv_w, conv_w.data(), conv_w.size() * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_conv_b, conv_b.data(), conv_b.size() * sizeof(float), hipMemcpyHostToDevice);

        hipEvent_t t0, t1;
        hipEventCreate(&t0);
        hipEventCreate(&t1);
        hipEventRecord(t0);

        float t_conv = 0, t_pool = 0, t_fc = 0, t_h2d = 0, t_d2h = 0;
        int correct = 0;
        std::ofstream cls("classification_results.csv");
        cls << "image_id,true_label,pred_label,confidence\n";

        for (int i = 0; i < n_images; ++i) {
            std::vector<float> in(Kin * H * W);
            for (int p = 0; p < H * W; ++p) in[p] = images_u8[i * H * W + p] / 255.0f;

            hipEvent_t eh2d0, eh2d1;
            hipEventCreate(&eh2d0); hipEventCreate(&eh2d1);
            hipEventRecord(eh2d0);
            hipMemcpy(d_input, in.data(), Kin * H * W * sizeof(float), hipMemcpyHostToDevice);
            hipEventRecord(eh2d1);
            hipEventSynchronize(eh2d1);
            float ms;
            hipEventElapsedTime(&ms, eh2d0, eh2d1);
            t_h2d += ms;

            // ---- Convolution ----
            hipEvent_t ec0, ec1;
            hipEventCreate(&ec0); hipEventCreate(&ec1);
            hipEventRecord(ec0);
            dim3 threads(16, 16);
            dim3 blocks((out_w + 15) / 16, (out_h + 15) / 16, Kout);
            conv_naive<<<blocks, threads>>>(d_input, d_conv_w, d_conv_b, d_conv_out, Kin, Kout, H, W, K);
            hipDeviceSynchronize();
            hipEventRecord(ec1);
            hipEventSynchronize(ec1);
            hipEventElapsedTime(&ms, ec0, ec1);
            t_conv += ms;

            // ---- Pooling ----
            hipEvent_t ep0, ep1;
            hipEventCreate(&ep0); hipEventCreate(&ep1);
            hipEventRecord(ep0);
            dim3 blockPool(256);
            dim3 gridPool((pool_h * pool_w + 255) / 256, Kout);
            maxpool_sample<<<gridPool, blockPool>>>(d_conv_out, d_pool, Kout, out_h, out_w, pool_k);
            hipDeviceSynchronize();
            hipEventRecord(ep1);
            hipEventSynchronize(ep1);
            hipEventElapsedTime(&ms, ep0, ep1);
            t_pool += ms;

            // ---- Copy pooled output to host ----
            hipEvent_t ed2h0, ed2h1;
            hipEventCreate(&ed2h0); hipEventCreate(&ed2h1);
            hipEventRecord(ed2h0);
            std::vector<float> h_pool(D);
            hipMemcpy(h_pool.data(), d_pool, D * sizeof(float), hipMemcpyDeviceToHost);
            hipEventRecord(ed2h1);
            hipEventSynchronize(ed2h1);
            hipEventElapsedTime(&ms, ed2h0, ed2h1);
            t_d2h += ms;

            // ---- Fully Connected ----
            auto t_fc_start = std::chrono::high_resolution_clock::now();
            std::vector<float> outv(out_dim);
            for (int o = 0; o < out_dim; ++o) {
                float s = fc_b[o];
                for (int d = 0; d < D; ++d) s += fc_w[o * D + d] * h_pool[d];
                outv[o] = s;
            }
            auto t_fc_end = std::chrono::high_resolution_clock::now();
            t_fc += std::chrono::duration<float, std::milli>(t_fc_end - t_fc_start).count();

            // ---- Classification ----
            auto best_it = std::max_element(outv.begin(), outv.end());
            int best = std::distance(outv.begin(), best_it);
            float confidence = *best_it;
            cls << i << "," << labels[i] << "," << best << "," << confidence << "\n";
            if (best == labels[i]) correct++;
        }

        hipEventRecord(t1);
        hipEventSynchronize(t1);
        float total_ms = 0;
        hipEventElapsedTime(&total_ms, t0, t1);
        double acc = 100.0 * correct / n_images;

        std::ofstream perf("pipeline2_perf.csv");
        perf << "total_ms,conv_ms,pool_ms,fc_ms,h2d_ms,d2h_ms,accuracy_percent,n_images,D,out_dim\n";
        perf << total_ms << "," << t_conv/n_images << "," << t_pool/n_images << "," 
             << t_fc/n_images << "," << t_h2d/n_images << "," << t_d2h/n_images << ","
             << acc << "," << n_images << "," << D << "," << out_dim << "\n";
        perf.close();
        cls.close();

        std::cout << "Pipeline GPU completed!\n";
        std::cout << "Accuracy: " << acc << "%\n";
        std::cout << "Avg Conv: " << t_conv/n_images << " ms | Avg Pool: " << t_pool/n_images
                  << " ms | Avg FC: " << t_fc/n_images << " ms\n";
        std::cout << "Avg H2D: " << t_h2d/n_images << " ms | Avg D2H: " << t_d2h/n_images << " ms\n";

        hipFree(d_input); hipFree(d_conv_out); hipFree(d_pool);
        hipFree(d_conv_w); hipFree(d_conv_b);
    }
    catch (const std::exception &e) {
        std::cerr << "Error: " << e.what() << std::endl;
        return 1;
    }
    return 0;
}
